#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include <stdio.h>
#include <conio.h>
#include <iostream>
#include <algorithm> 
#include <time.h>
#include <stdio.h>
using namespace std;

const int MAXITER = 100000;
const int BLOCK_SIZE = 32;
const int N = 32 * 32;					// ���������� ���������� ����� �����
const int SIZE = N + 2;					// ����� ���������� ����� � �����
float LENGTH = 10;						// ����� ��������� �������
const float h = LENGTH / (SIZE - 1);	// �������� ���� �����
const float a = 1.0;					// �������� ����. ���������
float F[SIZE][SIZE];					// ������� �������� ������� �� �������� �����

const float h_sq = h * h;
const float c = 4.0 / h_sq + a;

__constant__ float constants[3];

__device__
float r(float x, float y)
{
	return - (x + y);
}

__global__ void kernel(float* prev, float* current, int* end)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x + 1;
	int i = blockIdx.y * blockDim.x + threadIdx.y + 1;
	float Fi = (prev[(i - 1) * SIZE + j] + prev[(i + 1) * SIZE + j]) / constants[1];
	float Fj = (prev[i * SIZE + j - 1] + prev[i * SIZE + j + 1]) / constants[1];
	current[i * SIZE + j] = (Fi + Fj - r(i * constants[0], j * constants[0])) / constants[2];
	if (fabs(current[i * SIZE + j] - prev[i * SIZE + j]) > 1e-5)
	{
		end[0] = 1;
	}
}

__host__
float solution(float x, float y)
{
	return x + y;
}

__host__
void Init()
{
	int i, j;
	for (i = 0; i < SIZE; i++)
	{
		for (j = 0; j < SIZE; j++)
		{
			if ((i != 0) && (j != 0) && (i != SIZE - 1) && (j != SIZE - 1))
			{
				F[i][j] = 0;
			}
			else
			{
				F[i][j] = solution(i * h, j * h);
			}
		}
	}
}


int main(int argc, char * argv[])
{
	float * prev = NULL;
	float * current = NULL;
	int* end;
	clock_t start;
	double duration;
	
	int* complete = new int[1];
	Init();
	float * ar = new float[3];
	ar[0] = h;
	ar[1] = h_sq;
	ar[2] = c;
	hipMemcpyToSymbol(HIP_SYMBOL(constants), ar, 3 * sizeof(float));
	hipMalloc((void**)&prev, SIZE * SIZE * sizeof (float));
	hipMalloc((void**)&current, SIZE * SIZE * sizeof (float));
	hipMalloc((void**)&end, sizeof(int));

	hipMemcpy(prev, F, SIZE * SIZE * sizeof (float), hipMemcpyHostToDevice);
	hipMemcpy(current, F, SIZE * SIZE * sizeof (float), hipMemcpyHostToDevice);
	start = clock();
	
	int iteration = 1;
	do
	{
		hipMemset(end, 0, 1);
		kernel << <dim3(N / BLOCK_SIZE, N / BLOCK_SIZE, 1), dim3(BLOCK_SIZE, BLOCK_SIZE, 1) >> > (prev, current, end);
		hipMemcpy(complete, end, sizeof (int), hipMemcpyDeviceToHost);
		swap(prev, current);
		iteration++;

	} while (iteration < MAXITER);

	hipMemcpy(F, prev, SIZE * SIZE * sizeof (float), hipMemcpyDeviceToHost);
	hipFree(prev);
	float maxError = 0;
	for (int i = 0; i < SIZE; i++)
	{
		for (int j = 0; j < SIZE; j++)
		{
			if (fabs(F[i][j] - solution(i * h, j * h)) > maxError)
			{
				maxError = fabs(F[i][j] - solution(i * h, j * h));
			}
		}
	}

	duration = (clock() - start) / (double)CLOCKS_PER_SEC;

	std::cout << "time: " << duration << '\n';
	printf("Iterations = %d\nError = %f", iteration, maxError);
	return 0;
}